#include "hip/hip_runtime.h"

#include <stdio.h>
#include <time.h>
#include <math.h>

__global__ void par_min(float* input) {
	const int tid = threadIdx.x;
	int no_threads = blockDim.x;
	int step = 1;
	
	while (no_threads > 0) {
		if (tid < no_threads) {
			int i1 = tid * step * 2;
			int i2 = i1 + step;
			if (input[i1] > input[i2])
				input[i1] = input[i2];
		}
		no_threads >>= 1;
		step <<= 1;
	}
}

__global__ void par_max(float* input) {
	const int tid = threadIdx.x;
	int no_threads = blockDim.x;
	int step = 1;
	
	while (no_threads > 0) {
		if (tid < no_threads) {
			int i1 = tid * step * 2;
			int i2 = i1 + step;
			if (input[i1] < input[i2])
				input[i1] = input[i2];
		}
		step <<= 1;
		no_threads >>= 1;
	}
}

__global__ void par_sum(float* input) {
	const int tid = threadIdx.x;
	int no_threads = blockDim.x;
	int step = 1;
	
	while (no_threads > 0) {
		if (tid < no_threads) {
			int i1 = tid * step * 2;
			int i2 = i1 + step;
			input[i1] += input[i2];
		}
		step <<= 1;
		no_threads >>= 1;
	}
}

__global__ void par_std(float* input, float avg) {
	const int tid = threadIdx.x;
	int no_threads = blockDim.x;
	int step = 1;
	
	while (no_threads > 0) {
		if (tid < no_threads) {
			int i1 = tid * step * 2;
			int i2 = i1 + step;
			input[i1] = (input[i1] - avg) * (input[i1] - avg);
			input[i2] = (input[i2] - avg) * (input[i2] - avg);
			input[i1] += input[i2];
		}
		step <<= 1;
		no_threads >>= 1;
	}
}

int main() {
	srand(time(NULL));
	const int N = 1<<7;
	float *a, *dev_min, *dev_max, *dev_sum, *dev_std;
	const int size = N * sizeof(float);
	clock_t t;
	float result;
	
	a = (float*) malloc(size);
	printf("Array: of %d", N);
	for (int i = 0; i < N; i++) {
		a[i] = rand() % N + 1;
		printf("%f ", a[i]);
	}
	
	//-------------------Min----------------------
	hipMalloc(&dev_min, size);
	hipMemcpy(dev_min, a, size, hipMemcpyHostToDevice);
	t = clock();
	par_min<<<1, N/2>>>(dev_min);
	hipMemcpy(&result, dev_min, sizeof(float), hipMemcpyDeviceToHost);
	t = clock() - t;
	printf("\n\nMinimum value: %f\ttime taken: %f milliseconds\n", result, (1000 * (double) t / CLOCKS_PER_SEC));
	
	//-------------------Max----------------------
	hipMalloc(&dev_max, size);
	hipMemcpy(dev_max, a, size, hipMemcpyHostToDevice);
	t = clock();
	par_max<<<1, N/2>>>(dev_max);
	hipMemcpy(&result, dev_max, sizeof(float), hipMemcpyDeviceToHost);
	t = clock() - t;
	printf("\nMaximum value: %f\ttime taken: %f milliseconds\n", result, (1000 * (double) t / CLOCKS_PER_SEC));
	
	//-------------------Sum----------------------
	hipMalloc(&dev_sum, size);
	hipMemcpy(dev_sum, a, size, hipMemcpyHostToDevice);
	t = clock();
	par_sum<<<1, N/2>>>(dev_sum);
	hipMemcpy(&result, dev_sum, sizeof(float), hipMemcpyDeviceToHost);
	t = clock() - t;
	printf("\nSum: %f\ttime taken: %f milliseconds\n", result, (1000 * (double) t / CLOCKS_PER_SEC));
	
	//-------------------Avg----------------------
	hipMalloc(&dev_sum, size);
	hipMemcpy(dev_sum, a, size, hipMemcpyHostToDevice);
	t = clock();
	par_sum<<<1, N/2>>>(dev_sum);
	hipMemcpy(&result, dev_sum, sizeof(float), hipMemcpyDeviceToHost);
	result = result / N;
	t = clock() - t;
	printf("\nAverage: %f\ttime taken: %f milliseconds\n", result, (1000 * (double) t / CLOCKS_PER_SEC));
	
	//-------------------Std----------------------
	hipMalloc(&dev_std, size);
	hipMemcpy(dev_std, a, size, hipMemcpyHostToDevice);
	t = clock();
	par_std<<<1, N/2>>>(dev_std, result);
	hipMemcpy(&result, dev_std, sizeof(float), hipMemcpyDeviceToHost);
	result = sqrt(result / N);
	t = clock() - t;
	printf("\nStandard deviation: %f\ttime taken: %f milliseconds\n", result, (1000 * (double) t / CLOCKS_PER_SEC));
	
	// clean up
	hipFree(dev_min);
	hipFree(dev_max);
	hipFree(dev_sum);
	hipFree(dev_std);
	delete[] a;
	
	return 0;
}

